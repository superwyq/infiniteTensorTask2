#include "softmax.cuh"

void softmax_gpu(float *input, float *output, int M, int N)
{
    double st, ela;
    st = get_walltime();

    // auto start_cpu = std::chrono::high_resolution_clock::now();
    dim3 block_dim(BLOCK_DIM, 1);
    dim3 grid_dim(M, 1);
    float* d_input,*d_output;
    hipMalloc((void**)&d_input, M * N * sizeof(float));
    hipMalloc((void**)&d_output, M * N * sizeof(float));
    hipMemcpy(d_input, input, M * N * sizeof(float), hipMemcpyHostToDevice);
    
    
    hipEvent_t start, stop;
    float ker_time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    
    KERNEL_VERSION<<<grid_dim, block_dim>>>(d_input, d_output, M, N);

    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&ker_time, start, stop);

    hipMemcpy(output, d_output, M * N * sizeof(float), hipMemcpyDeviceToHost);
    hipFree(d_input);
    hipFree(d_output);
    
    // auto end_cpu = std::chrono::high_resolution_clock::now();
    // auto duration = std::chrono::duration_cast<std::chrono::milliseconds>(end_cpu - start_cpu);

    // std::cout << "total use time: " << duration.count() * 1.0 << std::endl;
    ela = get_walltime() - st;
    std::cout << "********** softmax_v1**********" <<std::endl; 
    std::cout << "Data size: " << M << " * " << N << std::endl;
    std::cout << "use time: " << ela << std::endl;
    std::cout << "kernel time: " << ker_time / 1000. << std::endl;
    printf("Bandwidth: %f GB/s\n", 4 * M * N * sizeof(float) * 1000.0 / (1<<30) / ker_time );
}